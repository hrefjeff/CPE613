#include "hip/hip_runtime.h"
#include <convolution.h>

static __device__ __host__ inline
hipfftComplex ComplexMul(hipfftComplex a, hipfftComplex b) {
    hipfftComplex c;
    c.x = a.x * b.x - a.y * b.y;
    c.y = a.x * b.y + a.y * b.x;
    return c;
}

__global__ void convolve_1d_time_kernel (
    float *input, 
    float *kernel,
    float *output,
    int N,
    int K
) {
    int idxInput = threadIdx.x + blockIdx.x * blockDim.x;
    if (idxInput > N + K - 1) return;

    float result = 0.0;
    for (int idxFilter = 0; idxFilter < K; idxFilter++) {
        if((idxInput - idxFilter) < 0 || (idxInput - idxFilter) >= N)
            result += 0;
        else
            result += (float)(kernel[idxFilter] * input[idxInput - idxFilter]);
    }
    output[idxInput] = result;
}

void convolve_1d (
    float* input,
    float* filter,
    float* output,
    int N,
    int K
){

    int numOfThreads = 32;
    int numOfBlocks = ((N + K - 1) + numOfThreads - 1) / numOfThreads;

    convolve_1d_time_kernel<<<numOfBlocks, numOfThreads>>> 
    (
        input,
        filter,
        output,
        N,
        K
    ); 
  
    checkCudaErrors(hipGetLastError());
}

__global__ void complexMulGPUKernel(
                    hipfftComplex* input1,
                    hipfftComplex* input2,
                    hipfftComplex* output,
                    int size
                ){
                            
    for (int idx = threadIdx.x + blockDim.x * blockIdx.x;
        idx < size;
        idx += blockDim.x * gridDim.x
    ){
        output[idx] = ComplexMul(input1[idx], input2[idx]);
    }
}

void complexMulGPU(
        hipfftComplex* input1,
        hipfftComplex* input2,
        hipfftComplex* output,
        int size 
    ) {
    int blockSize = 32;
    int gridSize = (size + blockSize - 1) / blockSize;

    complexMulGPUKernel<<<gridSize, blockSize>>>(output, input1, input2, size);

    checkCudaErrors(hipGetLastError());
}

bool read_file_into_vector(std::string filename, std::vector<float>& arr) {
    std::ifstream the_file(filename);

    if (the_file.is_open()) {
        int index = 0;
        float value;
        while (the_file >> value) {
            arr[index++] = (float)(value);
        }
        the_file.close();
    } else {
        std::cout << "Unable to open signal file." << std::endl;
        return false;
    }
    return true;
}

template <typename T>
void dataTypeWriter(FILE* filePtr);

template<>
void dataTypeWriter<double>(FILE* filePtr){
    fprintf(filePtr, "double\n");
}

template<>
void dataTypeWriter<hipfftComplex>(FILE* filePtr){
    fprintf(filePtr, "complex\n");
}

template<>
void typeSpecificfprintf<hipfftComplex>(FILE* fptr, hipfftComplex const & data){

    fprintf(fptr, "%20.16f %20.16f\n", data.x, data.y);

}

template<>
void typeSpecificfprintf<double>(FILE* fptr, double const & data){

    fprintf(fptr, "%20.16f\n", data);

}