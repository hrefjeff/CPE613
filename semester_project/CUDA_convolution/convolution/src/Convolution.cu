#include "hip/hip_runtime.h"
#include <Convolution.h>

/**
 * Constructor for Convolution class
 * @param sizeOfSignals: size of input signal(s)
 * @param numOfSignals: number of input signal(s)
*/
Convolution::Convolution(
    int sizeOfSignals,
    int numOfSignals
) : _signal_size(sizeOfSignals), _batch_size(numOfSignals) {
    if (sizeOfSignals <= 512) {
        _method = Method::TimeBased;
    } else {
        _method = Method::FFTBased;
        _fft_size = (sizeOfSignals + sizeOfSignals - 1);
    }
}

/***
 * Return pointer to host signal array
*/
float* Convolution::get_signal() {
    return _hf_signal;
}

/***
 * Return pointer to host filter array
*/
float* Convolution::get_filter() {
    return _hf_filter;
}

std::vector<hipfftComplex> Convolution::get_signal_complex() {
    return _hc_signal;
}

std::vector<hipfftComplex> Convolution::get_filter_complex() {
    return _hc_filter;
}

/***
 * Allocate memory for signal, filter, and output on host and device
*/
void Convolution::allocate_float_memory(){
    // Allocate memory for signal, filter, and output
    int total_size = _signal_size + _signal_size - 1;
    _hf_signal = new float[_signal_size];
    _hf_filter = new float[_signal_size];
    _hf_output = new float[total_size - 1];
    checkCudaErrors(
        hipMalloc((void **)&_df_signal, _signal_size * sizeof(float))
    );
    checkCudaErrors(
        hipMalloc((void **)&_df_filter, _signal_size * sizeof(float))
    );
    checkCudaErrors(
        hipMalloc((void **)&_df_output, total_size * sizeof(float))
    );
}

void Convolution::allocate_complex_memory() {
    _fft_size = next_power_of_two(_signal_size + _signal_size - 1);
    
    _hc_signal.resize(_fft_size, hipfftComplex{0});
    _hc_filter.resize(_fft_size, hipfftComplex{0});
    _hc_convolved_result.resize(_fft_size, hipfftComplex{0});

    checkCudaErrors(
        hipMalloc(
            (void **)&_dc_signal,
            _hc_signal.size() * sizeof(hipfftComplex)
        )
    );
    checkCudaErrors(
        hipMalloc(
            (void **)&_dc_filter,
            _hc_filter.size() * sizeof(hipfftComplex)
        )
    );
    checkCudaErrors(
        hipMalloc(
            (void **)&_dc_convolved_result,
            _hc_convolved_result.size() * sizeof(hipfftComplex)
        )
    );
}

/***
 * Read data from file and put it into array
 * @param filename: name of file to read from
 * @param host_arr: array to put data into
*/
void Convolution::read_file_into_array(
    std::string filename,
    float* host_arr
) {
    std::ifstream the_file(filename);

    if (the_file.is_open()) {
        int index = 0;
        float value;
        while (the_file >> value) {
            host_arr[index++] = (float)(value);
        }
        the_file.close();
    } else {
        std::cout << "Unable to open signal file." << std::endl;
    }
}

/***
* Reads a file into a signal vector of type hipfftComplex
* @param filename - the name of the file to read
***/
void Convolution::read_file_into_complex_signal(std::string filename) {
    std::ifstream the_file(filename);

    if (the_file.is_open()) {
        int index = 0;
        float value;
        while (the_file >> value) {
            _hc_signal[index++] = float_to_complex(value);
        }
        the_file.close();
    } else {
        std::cout << "Unable to open signal file." << std::endl;
    }
}

/***
* Reads a file into a filter vector of type hipfftComplex
* @param filename - the name of the file to read
* @param arr - the vector to read the file into
***/
void Convolution::read_file_into_complex_filter(std::string filename) {
    std::ifstream the_file(filename);

    if (the_file.is_open()) {
        int index = 0;
        float value;
        while (the_file >> value) {
            _hc_filter[index++] = float_to_complex(value);
        }
        the_file.close();
    } else {
        std::cout << "Unable to open signal file." << std::endl;
    }
}

/***
 * Write results to file
*/
void Convolution::write_results_to_file(const char* file_name) {
    checkCudaErrors(
        hipMemcpy(
            _hf_output, _df_output,
            (_signal_size + _signal_size - 1) * sizeof(float),
            hipMemcpyDeviceToHost
        )
    );
    FILE* filePtr = fopen(file_name, "w");
    for (int i = 0; i < _signal_size + _signal_size - 1; i++) {
        fprintf (filePtr, "%20.16e\n", _hf_output[i]);
    }
    fclose(filePtr);
}

/***
 * Write results to file
*/
void Convolution::write_complex_results_to_file(const char* file_name) {
    checkCudaErrors(
        hipMemcpy(
            _hc_convolved_result.data(), _dc_convolved_result,
            _fft_size * sizeof(hipfftComplex),
            hipMemcpyDeviceToHost
        )
    );

    FILE* filePtr = fopen(file_name, "w");
    for (int i = 0; i < _fft_size; i++) {
        fprintf(
            filePtr,
            "%20.16e\n",
            complex_to_float(_hc_convolved_result[i])
        );
    }
    fclose(filePtr);
}

/***
 * Compute the convolution
*/
void Convolution::compute(){
    if (_method == Method::TimeBased) {
        checkCudaErrors(
            hipMemcpy(
                _df_signal, _hf_signal,
                _signal_size * sizeof(float),
                hipMemcpyHostToDevice
            )
        );
        checkCudaErrors(
            hipMemcpy(
                _df_filter, _hf_filter,
                _signal_size * sizeof(float),
                hipMemcpyHostToDevice
            )
        );
        convolve_1d_time(
            _df_signal,
            _df_filter,
            _df_output,
            _signal_size,
            _signal_size
        );
    } else {
        hipfftHandle plan;
        hipfftCreate(&plan);
        hipfftPlan1d(&plan, _fft_size, HIPFFT_C2C, _batch_size);

        checkCudaErrors(
            hipMemcpy(
                _dc_signal, _hc_signal.data(),
                sizeof(hipfftComplex) * _hc_signal.size(),
                hipMemcpyHostToDevice
            )
        );

        checkCudaErrors(
            hipMemcpy(
                _dc_filter, _hc_filter.data(),
                sizeof(hipfftComplex) * _hc_filter.size(),
                hipMemcpyHostToDevice
            )
        );
        
        // Process signal    
        checkCudaErrors(
            hipfftExecC2C(plan, _dc_signal, _dc_signal, HIPFFT_FORWARD)
        );

        // Process filter
        checkCudaErrors(
            hipfftExecC2C(plan, _dc_filter, _dc_filter, HIPFFT_FORWARD)
        );

        checkCudaErrors(hipDeviceSynchronize());

        convolve_1d_fft(
            _dc_signal,
            _dc_filter,
            _dc_convolved_result,
            _fft_size
        );
        
        // Perform inverse to get result
        checkCudaErrors(
            hipfftExecC2C(
                plan, _dc_convolved_result, _dc_convolved_result, HIPFFT_BACKWARD
            )
        );
        
        hipfftDestroy(plan);
    }
}

/***
 * Find the next power of two above the number provided
*/
int Convolution::next_power_of_two(int num) {
    return 1 << (int(log2(num - 1)) + 1);
}

/***
 * Scale the value based on the size of the signal
*/
static __device__ __host__ inline
hipfftComplex ComplexScale(hipfftComplex a, float s) {
    hipfftComplex c;
    c.x = s * a.x;
    c.y = s * a.y;
    return c;
}

/***
 * Perform multiplication of two complex numbers
*/
static __device__ __host__ inline
hipfftComplex ComplexMul(hipfftComplex a, hipfftComplex b) {
    hipfftComplex c;
    c.x = a.x * b.x - a.y * b.y;
    c.y = a.x * b.y + a.y * b.x;
    return c;
}

/***
 * @brief: 1D convolution in time domain
 * @param input: input signal
 * @param kernel: filter
 * @param output: output signal
 * @param N: length of input signal
 * @param K: length of filter
*/
__global__
void convolve_1d_time_kernel (
    float *input, 
    float *kernel,
    float *output,
    int N,
    int K
) {
    int idxInput = threadIdx.x + blockIdx.x * blockDim.x;
    if (idxInput > N + K - 1) return;

    float result = 0.0;
    for (int idxFilter = 0; idxFilter < K; idxFilter++) {
        if((idxInput - idxFilter) < 0 || (idxInput - idxFilter) >= N)
            result += 0;
        else
            result += (float)(kernel[idxFilter] * input[idxInput - idxFilter]);
    }
    output[idxInput] = result;
}

/***
 * @brief: 1D convolution in frequency domain
 * @param input: input signal
 * @param kernel: filter
 * @param output: output signal
 * @param N: length of input signal
 * @param K: length of filter
*/
void Convolution::convolve_1d_time (
    float* input,
    float* filter,
    float* output,
    int N,
    int K
) {

    int numOfThreads = 1024;
    int numOfBlocks = ((N + K - 1) + numOfThreads - 1) / numOfThreads;

    convolve_1d_time_kernel<<<numOfBlocks, numOfThreads>>> 
    (
        input,
        filter,
        output,
        N,
        K
    ); 
  
    checkCudaErrors(hipGetLastError());
}

/***
 * @brief: 1D convolution in frequency domain
 * @param input: input signal
 * @param kernel: filter
 * @param output: output signal
 * @param size: size of FFT
*/
__global__
void complexMulGPUKernel(
    hipfftComplex* input1,
    hipfftComplex* input2,
    hipfftComplex* output,
    int size
) {
    for (int idx = threadIdx.x + blockDim.x * blockIdx.x;
        idx < size;
        idx += blockDim.x * gridDim.x
    ){
        output[idx] = ComplexScale(
                            ComplexMul(input1[idx], input2[idx]),
                            1.0 / size
                        );
    }
}

/***
 * Perform DFT on input signal
 * @param input: input signal
 * @param input2: filter kernel
 * @param output: output signal
 * @param size: size of FFT
*/
void Convolution::convolve_1d_fft(
        hipfftComplex* input1,
        hipfftComplex* input2,
        hipfftComplex* output,
        int size 
) {
    int blockSize = 1024;
    int gridSize = (size + blockSize - 1) / blockSize;

    complexMulGPUKernel<<<gridSize, blockSize>>>(input1, input2, output, size);

    checkCudaErrors(hipGetLastError());
}