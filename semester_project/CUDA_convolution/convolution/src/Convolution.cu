#include "hip/hip_runtime.h"
#include <Convolution.h>

/**
 * Constructor for Convolution class
 * @param sizeOfSignals: size of input signal(s)
 * @param numOfSignals: number of input signal(s)
*/
Convolution::Convolution(
    int sizeOfSignals,
    int numOfSignals
) : _signal_size(sizeOfSignals), _batch_size(numOfSignals) {
    if (sizeOfSignals <= 512) {
        _method = Method::TimeBased;
    } else {
        _method = Method::FFTBased;
    }
}

std::vector<hipfftComplex> Convolution::get_signal() {
    return _hc_signal;
}

std::vector<hipfftComplex> Convolution::get_filter() {
    return _hc_filter;
}

void Convolution::allocate_memory() {
    _output_size = next_power_of_two(_signal_size + _signal_size - 1);
    
    if (_method == Method::TimeBased) {
        _hc_signal.resize(_signal_size, hipfftComplex{0});
        _hc_filter.resize(_signal_size, hipfftComplex{0});
    } else if (_method == Method::FFTBased) {
        _hc_signal.resize(_output_size, hipfftComplex{0});
        _hc_filter.resize(_output_size, hipfftComplex{0});
    }
    
    _hc_output.resize(_output_size, hipfftComplex{0});

    checkCudaErrors(
        hipMalloc(
            (void **)&_dc_signal,
            _hc_signal.size() * sizeof(hipfftComplex)
        )
    );
    checkCudaErrors(
        hipMalloc(
            (void **)&_dc_filter,
            _hc_filter.size() * sizeof(hipfftComplex)
        )
    );
    checkCudaErrors(
        hipMalloc(
            (void **)&_dc_output,
            _hc_output.size() * sizeof(hipfftComplex)
        )
    );
}

/***
* Reads a file into a signal vector of type hipfftComplex
* @param filename - the name of the file to read
***/
void Convolution::read_file_into_signal(std::string filename) {
    std::ifstream the_file(filename);

    if (the_file.is_open()) {
        int index = 0;
        float value;
        while (the_file >> value) {
            _hc_signal[index++] = float_to_complex(value);
        }
        the_file.close();
    } else {
        std::cout << "Unable to open signal file." << std::endl;
    }
}

/***
* Reads a file into a filter vector of type hipfftComplex
* @param filename - the name of the file to read
***/
void Convolution::read_file_into_filter(std::string filename) {
    std::ifstream the_file(filename);

    if (the_file.is_open()) {
        int index = 0;
        float value;
        while (the_file >> value) {
            _hc_filter[index++] = float_to_complex(value);
        }
        the_file.close();
    } else {
        std::cout << "Unable to open signal file." << std::endl;
    }
    checkCudaErrors(
        hipMemcpy(
            _dc_filter, _hc_filter.data(),
            _hc_filter.size() * sizeof(hipfftComplex),
            hipMemcpyHostToDevice
        )
    );
}

/***
 * Write results to file
*/
void Convolution::write_results_to_file(const char* file_name) {
    checkCudaErrors(
        hipMemcpy(
            _hc_output.data(), _dc_output,
            _hc_output.size() - 1 * sizeof(hipfftComplex),
            hipMemcpyDeviceToHost
        )
    );

    FILE* filePtr = fopen(file_name, "w");
    for (int i = 0; i < _hc_output.size() - 1; i++) {
        fprintf(
            filePtr,
            "%20.16e\n",
            complex_to_float(_hc_output[i])
        );
    }
    fclose(filePtr);
}

/***
 * Compute the convolution
*/
void Convolution::compute(){
    if (_method == Method::TimeBased) {
        checkCudaErrors(
            hipMemcpy(
                _dc_signal, _hc_signal.data(),
                _signal_size * sizeof(hipfftComplex),
                hipMemcpyHostToDevice
            )
        );
        checkCudaErrors(
            hipMemcpy(
                _dc_filter, _hc_filter.data(),
                _signal_size * sizeof(hipfftComplex),
                hipMemcpyHostToDevice
            )
        );
        convolve_1d_time(
            _dc_signal,
            _dc_filter,
            _dc_output,
            _signal_size,
            _signal_size
        );
    } else {
        
        hipfftHandle plan;
        hipfftCreate(&plan);
        hipfftPlan1d(&plan, _output_size, HIPFFT_C2C, _batch_size);

        checkCudaErrors(
            hipMemcpy(
                _dc_signal, _hc_signal.data(),
                _hc_signal.size() * sizeof(hipfftComplex),
                hipMemcpyHostToDevice
            )
        );
        checkCudaErrors(
            hipMemcpy(
                _dc_filter, _hc_filter.data(),
                _hc_filter.size() * sizeof(hipfftComplex),
                hipMemcpyHostToDevice
            )
        );
        
        // Process signal    
        checkCudaErrors(
            hipfftExecC2C(plan, _dc_signal, _dc_signal, HIPFFT_FORWARD)
        );

        // Process filter
        checkCudaErrors(
            hipfftExecC2C(plan, _dc_filter, _dc_filter, HIPFFT_FORWARD)
        );

        checkCudaErrors(hipDeviceSynchronize());

        convolve_1d_fft(
            _dc_signal,
            _dc_filter,
            _dc_output,
            _output_size
        );
        
        // Perform inverse to get result
        checkCudaErrors(
            hipfftExecC2C(
                plan, _dc_output, _dc_output, HIPFFT_BACKWARD
            )
        );
        
        hipfftDestroy(plan);
    }
}

/***
 * Find the next power of two above the number provided
*/
int Convolution::next_power_of_two(int num) {
    return 1 << (int(log2(num - 1)) + 1);
}

/***
 * Scale the value based on the size of the signal
*/
static __device__ __host__ inline
hipfftComplex ComplexScale(hipfftComplex a, float s) {
    hipfftComplex c;
    c.x = s * a.x;
    c.y = s * a.y;
    return c;
}

/***
 * Perform multiplication of two complex numbers
*/
static __device__ __host__ inline
hipfftComplex ComplexMul(hipfftComplex a, hipfftComplex b) {
    hipfftComplex c;
    c.x = a.x * b.x - a.y * b.y;
    c.y = a.x * b.y + a.y * b.x;
    return c;
}

// Complex addition
static __device__ __host__ inline
hipfftComplex ComplexAdd(hipfftComplex a, hipfftComplex b) {
  hipfftComplex c;
  c.x = a.x + b.x;
  c.y = a.y + b.y;
  return c;
}

/***
 * @brief: 1D convolution in time domain
 * @param input: input signal
 * @param kernel: filter
 * @param output: output signal
 * @param N: length of input signal
 * @param K: length of filter
*/
__global__
void convolve_1d_time_kernel (
    hipfftComplex *input, 
    hipfftComplex *kernel,
    hipfftComplex *output,
    int N,
    int K
) {
    int idxInput = threadIdx.x + blockIdx.x * blockDim.x;
    if (idxInput > N + K - 1) return;

    hipfftComplex result = hipfftComplex{0.0};
    for (int idxFilter = 0; idxFilter < K; idxFilter++) {
        if((idxInput - idxFilter) < 0 || (idxInput - idxFilter) >= N) {
            result = ComplexAdd(result, hipfftComplex{0.0});
        }
        else {
            result = ComplexAdd(
                        result,
                        ComplexMul(
                            kernel[idxFilter],
                            input[idxInput - idxFilter]
                        )
                );
        }
    }
    output[idxInput] = result;
}

/***
 * @brief: 1D convolution in frequency domain
 * @param input: input signal
 * @param kernel: filter
 * @param output: output signal
 * @param N: length of input signal
 * @param K: length of filter
*/
void Convolution::convolve_1d_time (
    hipfftComplex* input,
    hipfftComplex* filter,
    hipfftComplex* output,
    int N,
    int K
) {

    int numOfThreads = 1024;
    int numOfBlocks = ((N + K - 1) + numOfThreads - 1) / numOfThreads;

    convolve_1d_time_kernel<<<numOfBlocks, numOfThreads>>> 
    (
        input,
        filter,
        output,
        N,
        K
    );
  
    checkCudaErrors(hipGetLastError());
}

/***
 * @brief: 1D convolution in frequency domain
 * @param input: input signal
 * @param kernel: filter
 * @param output: output signal
 * @param size: size of FFT
*/
__global__
void complexMulGPUKernel(
    hipfftComplex* input1,
    hipfftComplex* input2,
    hipfftComplex* output,
    int size
) {
    for (int idx = threadIdx.x + blockDim.x * blockIdx.x;
        idx < size;
        idx += blockDim.x * gridDim.x
    ){
        output[idx] = ComplexScale(
                        ComplexMul(input1[idx], input2[idx]),
                        1.0 / size
                    );
    }
}

/***
 * Perform DFT on input signal
 * @param input: input signal
 * @param input2: filter kernel
 * @param output: output signal
 * @param size: size of FFT
*/
void Convolution::convolve_1d_fft(
        hipfftComplex* input1,
        hipfftComplex* input2,
        hipfftComplex* output,
        int size 
) {
    int blockSize = 1024;
    int gridSize = (size + blockSize - 1) / blockSize;

    complexMulGPUKernel<<<gridSize, blockSize>>>(input1, input2, output, size);

    checkCudaErrors(hipGetLastError());
}


/***
 * Dump GPU data to a file
 * @param devicePtrToData: pointer to data on GPU
 * @param dimensionsOfData: dimensions of data
 * @param filename: name of file to dump data to
*/
template<typename T>
void dumpGPUDataToFile(
    T* devicePtrToData,
    std::vector<int> dimensionsOfData,
    std::string filename
) {

    //checkCudaErrors(hipDeviceSynchronize()); // force GPU thread to wait

    int totalNumElements = 1;
    for(auto elts : dimensionsOfData) {
        totalNumElements *= elts;
    }

    std::vector<T> hostData(totalNumElements, T{0});

    checkCudaErrors(hipMemcpy(
        hostData.data(),
        devicePtrToData,
        totalNumElements * sizeof(T),
        hipMemcpyDeviceToHost
    ));


    // size of vector of dims
    FILE* filePtr = fopen(filename.c_str(), "w");
    // write how many dims we have
    fprintf(filePtr, "%zu\n", dimensionsOfData.size());
    for(auto elts : dimensionsOfData) {
        fprintf(filePtr,"%d\n", elts);
    }

    dataTypeWriter<T>(filePtr);

    for(auto elt : hostData) {
        // support multiple types or use C++
        typeSpecificfprintf(filePtr, elt);
    }
    fclose(filePtr);
}


template <typename T>
void dataTypeWriter(FILE* filePtr);

template<>
void dataTypeWriter<double>(FILE* filePtr){
    fprintf(filePtr, "double\n");
}

template<>
void dataTypeWriter<hipfftComplex>(FILE* filePtr){
    fprintf(filePtr, "complex\n");
}

template<>
void dataTypeWriter<float>(FILE* filePtr){
    fprintf(filePtr, "float\n");
}

template<>
void typeSpecificfprintf<hipfftComplex>(FILE* fptr, hipfftComplex const & data){

    fprintf(fptr, "%20.16e %20.16e\n", data.x, data.y);

}

template<>
void typeSpecificfprintf<double>(FILE* fptr, double const & data){

    fprintf(fptr, "%20.16f\n", data);

}

template<>
void typeSpecificfprintf<float>(FILE* fptr, float const & data){

    fprintf(fptr, "%20.16e\n", data);

}