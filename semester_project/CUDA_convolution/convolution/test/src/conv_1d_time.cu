/*
    Testing for 1D Time Domain Convolution
    To compile: nvcc test.cu -o test.o -g -G
    To debug: cuda-gdb test.o
    Useful debug tools:
        set cuda coalescing off
        break main
        break 28
        run
        continue
        info cuda threads
        print result
*/

/* Include C++ stuff */
#include <iostream>
#include <string.h>
#include <iostream>
#include <complex>

#include <convolution.h>
#include <Timer.hpp>

#define N 8192
#define K 8192

using namespace std;

int main() {
    float *h_input = new float[N];
    float *h_filter = new float[K];
    float *h_output = new float[N + K - 1];
    float *d_input, *d_filter, *d_output;
    checkCudaErrors(hipMalloc((void **)&d_input, N * sizeof(float)));
    checkCudaErrors(hipMalloc((void **)&d_filter, K * sizeof(float)));
    checkCudaErrors(hipMalloc((void **)&d_output, (N+K-1) * sizeof(float)));

    // Prepare to read signal and filter information from files
    string signal_file_name =
        "/home/jeff/code/CPE613/semester_project/test_data_gold/arr1_8192.txt";
    string filter_file_name =
        "/home/jeff/code/CPE613/semester_project/test_data_gold/arr2_8192.txt";
    const char *output_file_name =
        "/home/jeff/code/CPE613/semester_project/test_data/cuda_time_8192.txt";

    ifstream signal_file(signal_file_name);
    ifstream filter_file(filter_file_name);

    if (signal_file.is_open()) {
        int index = 0;
        float value;
        while (signal_file >> value) {
            h_input[index++] = (float)(value);
        }
        signal_file.close();
    } else {
        std::cout << "Unable to open signal file." << std::endl;
        return EXIT_FAILURE;
    }

    if (filter_file.is_open()) {
        int index = 0;
        float value;
        while (filter_file >> value) {
            h_filter[index++] = (float)(value);
        }
        filter_file.close();
    } else {
        std::cout << "Unable to open filter file." << std::endl;
        return EXIT_FAILURE;
    }

    
    checkCudaErrors(
        hipMemcpy(
            d_input, h_input,
            N * sizeof(float),
            hipMemcpyHostToDevice
        )
    );
    checkCudaErrors(
        hipMemcpy(
            d_filter, h_filter,
            K * sizeof(float),
            hipMemcpyHostToDevice
        )
    );

    Timer timer;
    timer.start();
    convolve_1d_time(d_input, d_filter, d_output, N, K);
    timer.stop();
    
    checkCudaErrors(
        hipMemcpy(
            h_output, d_output,
            (N + K - 1) * sizeof(float),
            hipMemcpyDeviceToHost
        )
    );

    double elapsedTime_ms = timer.elapsedTime_ms();

    printf (
    "\n- Elapsed Time:             %20.16e Ms\n\n",
        elapsedTime_ms / 1.0e3
    );

    FILE* filePtr = fopen(output_file_name, "w");
    for (int i = 0; i < N + K - 1; i++) {
        fprintf (filePtr, "%20.16e\n", h_output[i]);
    }
    fclose(filePtr);

    delete[] h_input;
    delete[] h_filter;
    hipFree(d_input);
    hipFree(d_filter);
    hipFree(d_output);
    return 0;
}
