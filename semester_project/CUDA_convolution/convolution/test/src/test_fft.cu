/* TODO: Implement Callbacks

https://developer.nvidia.com/blog/cuda-pro-tip-use-cufft-callbacks-custom-data-processing/


*/

/* Include C++ stuff */
#include <complex>
#include <string.h>
#include <cstdio>
#include <cstdlib>

/* Include my stuff */
#include <convolution.h>
#include <Timer.hpp>

/* Include CUDA stuff */
#include <hip/hip_runtime.h>
#include <hipfft/hipfftXt.h>

#define N 1024
#define K 1024
#define BATCH_SIZE 1

using namespace std;

int main() {
    hipfftHandle plan1; // Forward FFT Plan
    hipfftHandle plan2; // Inverse FFT Plan
    hipStream_t stream = NULL;
    
    bool file_status = false;
    string signal_file_name =
        "/home/jeff/code/CPE613/semester_project/test_data_gold/arr1_1024.txt";
    string filter_file_name =
        "/home/jeff/code/CPE613/semester_project/test_data_gold/arr2_1024.txt";
    const char *output_file_name =
        "/home/jeff/code/CPE613/semester_project/test_data/cuda_fft_1024.txt";

    // Initialize the signal
    vector<hipfftComplex> h_signal(N);
    vector<hipfftComplex> h_signal_fft(N + K - 1);
    
    file_status = read_file_into_vector(signal_file_name, h_signal);
    if (file_status == false) return EXIT_FAILURE;

    hipfftComplex *d_signal = nullptr;
    hipfftComplex *d_signal_fft = nullptr;
    hipMalloc(reinterpret_cast<void **>(&d_signal),
                sizeof(hipfftComplex) * h_signal.size());
    hipMalloc(reinterpret_cast<void **>(&d_signal_fft),
                sizeof(hipfftComplex) * h_signal_fft.size());

    // Initialize the filter
    vector<hipfftComplex> h_filter(K);
    vector<hipfftComplex> h_filter_fft(N + K - 1);

    file_status = read_file_into_vector(filter_file_name, h_filter);
    if (file_status == false) return EXIT_FAILURE;
    
    hipfftComplex *d_filter = nullptr;
    hipfftComplex *d_filter_fft = nullptr;
    hipMalloc(reinterpret_cast<void **>(&d_filter),
                sizeof(hipfftComplex) * h_filter.size());
    hipMalloc(reinterpret_cast<void **>(&d_filter_fft),
                sizeof(hipfftComplex) * h_filter_fft.size());

    // Initialize the product
    vector<hipfftComplex> h_product_fft(N + K - 1);
    hipfftComplex *d_product_fft = nullptr;
    hipMalloc(reinterpret_cast<void **>(&d_product_fft),
                sizeof(hipfftComplex) * N + K - 1);

    vector<hipfftComplex> h_result(N + K - 1);

    checkCudaErrors(
        hipMemcpyAsync(
            d_signal, h_signal.data(),
            sizeof(hipfftComplex) * h_signal.size(),
            hipMemcpyHostToDevice,
            stream
        )
    );

    checkCudaErrors(
        hipMemcpyAsync(
            d_filter, h_filter.data(),
            sizeof(hipfftComplex) * h_filter.size(),
            hipMemcpyHostToDevice,
            stream
        )
    );

    hipfftCreate(&plan1);
    hipfftPlan1d(&plan1, N + K - 1, HIPFFT_C2C, BATCH_SIZE);
    hipStreamCreateWithFlags(&stream, hipStreamNonBlocking);
    hipfftSetStream(plan1, stream);

    // Process signal    
    hipfftExecC2C(plan1, d_signal, d_signal_fft, HIPFFT_FORWARD);

    // Process filter
    hipfftExecC2C(plan1, d_filter, d_filter_fft, HIPFFT_FORWARD);

    hipStreamSynchronize(stream); // force CPU thread to wait

    checkCudaErrors(hipGetLastError());

    dumpGPUDataToFile(d_signal_fft, {N + K - 1,1}, "cuda-fft-signal.txt");
    dumpGPUDataToFile(d_filter_fft, {N + K - 1,1}, "cuda-fft-filter.txt");

    // hipMemcpyAsync(h_signal_fft.data(), d_signal, sizeof(hipfftComplex) * N,
    //                              hipMemcpyDeviceToHost, stream);
    // hipMemcpyAsync(h_filter_fft.data(), d_filter, sizeof(hipfftComplex) * K,
    //                              hipMemcpyDeviceToHost, stream);

    // std::printf("Host signal fft array:\n");
    // for (int i = 0; i < 20; i++) {
    //     std::printf("%f + %fj\n", h_signal_fft[i].x, h_signal_fft[i].y);
    // }
    // std::printf("=====\n");

    // Multiplication section
    complexMulGPU(
        d_signal,
        d_filter,
        d_product_fft,
        N + K - 1
    );

    // printf("Host product fft:\n");
    // int z = 0;
    // for (auto &i : h_product_fft) {
    //     printf("%d : %f\n", z++, i.x);
    // }
    // printf("=====\n");

    // hipMemcpyAsync(d_product_fft, h_product_fft.data(), sizeof(hipfftComplex) * N,
    //                              hipMemcpyHostToDevice, stream);

    // dumpGPUDataToFile(d_product_fft, {N,1}, "test3.txt");

    // Perform inverse
    hipfftCreate(&plan2);
    hipfftPlan1d(&plan2, h_product_fft.size(), HIPFFT_C2C, BATCH_SIZE);

    // Execute the inverse FFT on the result
    hipfftExecC2C(plan2, d_product_fft, d_product_fft, HIPFFT_BACKWARD);

    hipStreamSynchronize(stream); // force CPU thread to wait
    
    hipMemcpyAsync(
        h_result.data(), d_product_fft,
        sizeof(hipfftComplex) * N + K - 1,
        hipMemcpyDeviceToHost,
        stream
    );

    //dumpGPUDataToFile(d_product_fft, {N,1}, output_file_name);

    hipStreamSynchronize(stream); // force CPU thread to wait

    FILE* filePtr = fopen(output_file_name, "w");
    float tmp;
    for(auto elt : h_result) {
        tmp = complex_to_float(elt);
        // support multiple types or use C++
        typeSpecificfprintf(filePtr, tmp);
    }
    fclose(filePtr);
    
    /* free resources */
    hipFree(d_signal);
    hipFree(d_signal_fft);
    hipFree(d_filter);
    hipFree(d_filter_fft);
    hipFree(d_product_fft);

    hipfftDestroy(plan1);
    hipfftDestroy(plan2);
    hipStreamDestroy(stream);
    
    

    hipDeviceReset();

    return EXIT_SUCCESS;
}