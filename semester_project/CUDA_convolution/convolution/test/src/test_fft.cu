/* Include C++ stuff */
#include <complex>
#include <string.h>
#include <cstdio>
#include <cstdlib>

/* Include my stuff */
#include <convolution.h>
#include <Timer.hpp>

/* Include CUDA stuff */
#include <hip/hip_runtime.h>
#include <hipfft/hipfftXt.h>

#define N 1024
#define K 1024
#define BATCH_SIZE 1

using namespace std;

int main() {
    hipfftHandle plan1;
    hipfftHandle plan2;
    hipfftHandle plan3;
    hipStream_t stream = NULL;
    
    bool file_status = false;
    string signal_file_name =
        "/home/jeff/code/CPE613/semester_project/test_data_gold/arr1_1024.txt";
    string filter_file_name =
        "/home/jeff/code/CPE613/semester_project/test_data_gold/arr2_1024.txt";
    const char *output_file_name =
        "/home/jeff/code/CPE613/semester_project/test_data/cuda_fft_1024.txt";

    // Initialize the signal
    vector<hipfftComplex> h_signal(N);
    vector<hipfftComplex> h_signal_fft(N);
    
    file_status = read_file_into_vector(signal_file_name, h_signal);
    if (file_status == false) return EXIT_FAILURE;

    hipfftComplex *d_signal = nullptr;
    hipMalloc(reinterpret_cast<void **>(&d_signal),
                sizeof(hipfftComplex) * h_signal.size());

    // Initial the filter
    vector<hipfftComplex> h_filter(K);

    file_status = read_file_into_vector(filter_file_name, h_filter);
    if (file_status == false) return EXIT_FAILURE;
    
    hipfftComplex *d_filter = nullptr;

    hipMalloc(reinterpret_cast<void **>(&d_filter),
                sizeof(hipfftComplex) * h_filter.size());

    // Initial the product
    vector<hipfftComplex> h_product_fft(N);
    hipfftComplex *d_product_fft = nullptr;
    hipMalloc(reinterpret_cast<void **>(&d_product_fft),
                sizeof(hipfftComplex) * N);

    vector<hipfftComplex> h_result(N);

    // printf("Signal array:\n");
    // int x = 0;
    // for (auto &i : h_signal) {
    //     printf("%d : %f\n", x++, i.x);
    // }
    // printf("=====\n");

    // printf("Filter array:\n");
    // int y = 0;
    // for (auto &i : h_filter) {
    //     printf("%d : %f\n", y++, i);
    // }
    // printf("=====\n");

    checkCudaErrors(
        hipMemcpyAsync(
            d_signal, h_signal.data(),
            sizeof(hipfftComplex) * h_signal.size(),
            hipMemcpyHostToDevice,
            stream
        )
    );

    checkCudaErrors(
        hipMemcpyAsync(
            d_filter, h_filter.data(),
            sizeof(hipfftComplex) * h_filter.size(),
            hipMemcpyHostToDevice,
            stream
        )
    );

    hipfftCreate(&plan1);
    hipfftPlan1d(&plan1, h_signal.size(), HIPFFT_C2C, BATCH_SIZE);

    hipStreamCreateWithFlags(&stream, hipStreamNonBlocking);
    hipfftSetStream(plan1, stream);
    
    // Perform transformation in place since idata&odata are the same
    hipfftExecC2C(plan1, d_signal, d_signal, HIPFFT_FORWARD);

    hipfftCreate(&plan2);
    hipfftPlan1d(&plan2, h_filter.size(), HIPFFT_C2C, BATCH_SIZE);

    hipStreamCreateWithFlags(&stream, hipStreamNonBlocking);
    hipfftSetStream(plan2, stream);

    hipfftExecC2C(plan2, d_filter, d_filter, HIPFFT_FORWARD);
    hipfftDestroy(plan2);

    hipStreamSynchronize(stream); // force CPU thread to wait

    // hipMemcpyAsync(h_signal_fft.data(), d_signal, sizeof(hipfftComplex) * N,
    //                              hipMemcpyDeviceToHost, stream);
    // hipMemcpyAsync(h_filter_fft.data(), d_filter, sizeof(hipfftComplex) * K,
    //                              hipMemcpyDeviceToHost, stream);

    // std::printf("Host signal fft array:\n");
    // for (int i = 0; i < 20; i++) {
    //     std::printf("%f + %fj\n", h_signal_fft[i].x, h_signal_fft[i].y);
    // }
    // std::printf("=====\n");

    // Multiplication section
    complexMulGPU(
        d_signal,
        d_filter,
        d_product_fft,
        N
    );

    // printf("Host product fft:\n");
    // int z = 0;
    // for (auto &i : h_product_fft) {
    //     printf("%d : %f\n", z++, i.x);
    // }
    // printf("=====\n");

    // hipMemcpyAsync(d_product_fft, h_product_fft.data(), sizeof(hipfftComplex) * N,
    //                              hipMemcpyHostToDevice, stream);

    // dumpGPUDataToFile(d_product_fft, {N,1}, "test3.txt");

    // Perform inverse
    hipfftCreate(&plan3);
    hipfftPlan1d(&plan3, h_product_fft.size(), HIPFFT_C2C, BATCH_SIZE);

    // Execute the inverse FFT on the result
    hipfftExecC2C(plan3, d_product_fft, d_product_fft, HIPFFT_BACKWARD);

    hipStreamSynchronize(stream); // force CPU thread to wait
    
    hipMemcpyAsync(h_result.data(), d_product_fft, sizeof(hipfftComplex) * N,
                                 hipMemcpyDeviceToHost, stream);

    //dumpGPUDataToFile(d_product_fft, {N,1}, output_file_name);

    FILE* filePtr = fopen(output_file_name, "w");
    float tmp;
    for(auto elt : h_result) {
        tmp = complex_to_float(elt);
        // support multiple types or use C++
        typeSpecificfprintf(filePtr, tmp);
    }
    fclose(filePtr);
    
    /* free resources */
    hipFree(d_signal);
    hipFree(d_filter);
    hipFree(d_product_fft);

    hipStreamDestroy(stream);

    hipDeviceReset();

    return EXIT_SUCCESS;
}