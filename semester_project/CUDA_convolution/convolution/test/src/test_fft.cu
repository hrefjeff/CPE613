#include "hip/hip_runtime.h"
/* Include C++ stuff */
#include <complex>
#include <string.h>
#include <cstdio>
#include <cstdlib>

/* Include my stuff */
#include <convolution.h>
#include <Timer.hpp>

/* Include CUDA stuff */
#include <hip/hip_runtime.h>
#include <hipfft/hipfftXt.h>

#define N 1024
#define K 1024
#define BATCH_SIZE 1

using namespace std;

// Complex data type
typedef float2 Complex;
static __device__ __host__ inline Complex ComplexMul(Complex, Complex);
void multiply_arrays_elementwise(const hipfftComplex* array1,
                                 const hipfftComplex* array2,
                                 vector<hipfftComplex> & result,
                                 int length
                                );

int main() {
    hipfftHandle plan1;
    hipfftHandle plan2;
    hipfftHandle plan3;
    hipStream_t stream = NULL;
    
    bool file_status = false;
    string signal_file_name =
        "/home/jeff/code/CPE613/semester_project/test_data_gold/arr1_1024.txt";
    string filter_file_name =
        "/home/jeff/code/CPE613/semester_project/test_data_gold/arr2_1024.txt";
    const char *output_file_name =
        "/home/jeff/code/CPE613/semester_project/test_data/cuda_fft_1024.txt";

    // Initialize the signal
    vector<hipfftComplex> h_signal(N);
    vector<hipfftComplex> h_signal_fft(N);
    
    file_status = read_file_into_vector(signal_file_name, h_signal);
    if (file_status == false) return EXIT_FAILURE;

    hipfftComplex *d_signal = nullptr;
    hipfftComplex *d_signal_fft = nullptr;

    hipMalloc(reinterpret_cast<void **>(&d_signal),
                sizeof(hipfftComplex) * h_signal.size());
    hipMalloc(reinterpret_cast<void **>(&d_signal_fft),
                sizeof(hipfftComplex) * h_signal_fft.size());

    // Initial the filter
    vector<hipfftComplex> h_filter(K);
    vector<hipfftComplex> h_filter_fft(K);

    file_status = read_file_into_vector(filter_file_name, h_filter);
    if (file_status == false) return EXIT_FAILURE;
    
    hipfftComplex *d_filter = nullptr;
    hipfftComplex *d_filter_fft = nullptr;

    hipMalloc(reinterpret_cast<void **>(&d_filter),
                sizeof(hipfftComplex) * h_filter.size());
    hipMalloc(reinterpret_cast<void **>(&d_filter_fft),
                sizeof(hipfftComplex) * h_filter_fft.size());

    // Initial the product
    vector<hipfftComplex> h_product_fft(N);
    hipfftComplex *d_product_fft = nullptr;
    hipMalloc(reinterpret_cast<void **>(&d_product_fft),
                sizeof(hipfftComplex) * N);

    vector<hipfftComplex> h_result(N);
    hipfftComplex* d_result = nullptr;
    hipMalloc(reinterpret_cast<void **>(&d_result), sizeof(hipfftComplex) * N);

    // printf("Signal array:\n");
    // int x = 0;
    // for (auto &i : h_signal) {
    //     printf("%d : %f\n", x++, i.x);
    // }
    // printf("=====\n");

    // printf("Filter array:\n");
    // int y = 0;
    // for (auto &i : h_filter) {
    //     printf("%d : %f\n", y++, i);
    // }
    // printf("=====\n");

    checkCudaErrors(
        hipMemcpyAsync(
            d_signal, h_signal.data(),
            sizeof(hipfftComplex) * h_signal.size(),
            hipMemcpyHostToDevice,
            stream
        )
    );

    checkCudaErrors(
        hipMemcpyAsync(
            d_filter, h_filter.data(),
            sizeof(hipfftComplex) * h_filter.size(),
            hipMemcpyHostToDevice,
            stream
        )
    );

    hipfftCreate(&plan1);
    hipfftPlan1d(&plan1, h_signal.size(), HIPFFT_C2C, BATCH_SIZE);

    hipStreamCreateWithFlags(&stream, hipStreamNonBlocking);
    hipfftSetStream(plan1, stream);
    
    hipfftExecC2C(plan1, d_signal, d_signal, HIPFFT_FORWARD);

    hipfftCreate(&plan2);
    hipfftPlan1d(&plan2, h_filter.size(), HIPFFT_C2C, BATCH_SIZE);

    hipStreamCreateWithFlags(&stream, hipStreamNonBlocking);
    hipfftSetStream(plan2, stream);

    hipfftExecC2C(plan2, d_filter, d_filter, HIPFFT_FORWARD);
    hipfftDestroy(plan2);

    hipStreamSynchronize(stream); // force CPU thread to wait

    hipMemcpyAsync(h_signal_fft.data(), d_signal, sizeof(hipfftComplex) * N,
                                 hipMemcpyDeviceToHost, stream);
    hipMemcpyAsync(h_filter_fft.data(), d_filter, sizeof(hipfftComplex) * K,
                                 hipMemcpyDeviceToHost, stream);

    // std::printf("Host signal fft array:\n");
    // for (int i = 0; i < 5; i++) {
    //     std::printf("%f + %fj\n", h_signal_fft[i].x, h_signal_fft[i].y);
    // }
    // std::printf("=====\n");

    // Multiplication section
    // complexMulGPU(
    //     d_signal_fft,
    //     d_filter_fft,
    //     d_product_fft,
    //     N
    // );

    multiply_arrays_elementwise(h_signal_fft.data(),
                                h_filter_fft.data(),
                                h_product_fft, 
                                N);

    // printf("Host product fft:\n");
    // int z = 0;
    // for (auto &i : h_product_fft) {
    //     printf("%d : %f\n", z++, i.x);
    // }
    // printf("=====\n");

    hipMemcpyAsync(d_product_fft, h_product_fft.data(), sizeof(hipfftComplex) * N,
                                 hipMemcpyHostToDevice, stream);

    // dumpGPUDataToFile(d_product_fft, {N,1}, "test3.txt");

    // Perform inverse
    hipfftCreate(&plan3);
    hipfftPlan1d(&plan3, h_product_fft.size(), HIPFFT_C2C, BATCH_SIZE);

    // Execute the inverse FFT on the result
    hipfftExecC2C(plan3, d_product_fft, d_result, HIPFFT_BACKWARD);

    hipStreamSynchronize(stream); // force CPU thread to wait
    
    hipMemcpyAsync(h_result.data(), d_result, sizeof(hipfftComplex) * N,
                                 hipMemcpyDeviceToHost, stream);

    dumpGPUDataToFile(d_result, {N,1}, output_file_name);
    
    /* free resources */
    hipFree(d_signal);
    hipFree(d_signal_fft);
    hipFree(d_filter);
    hipFree(d_filter_fft);
    hipFree(d_result);
    hipFree(d_product_fft);

    hipStreamDestroy(stream);

    hipDeviceReset();

    return EXIT_SUCCESS;
}

hipfftComplex float_to_complex(float value) {
    hipfftComplex complex_value;
    complex_value.x = value;  // Assign the float value to the real part
    complex_value.y = 0.0f;    // Set the imaginary part to zero
    return complex_value;
}

void multiply_arrays_elementwise(const hipfftComplex* array1,
                                 const hipfftComplex* array2,
                                 vector<hipfftComplex> & result,
                                 int length
                                ) {
    for (int i = 0; i < length; ++i) {
        result[i] = ComplexMul(array1[i], array2[i]);
    }
}

////////////////////////////////////////////////////////////////////////////////
// Complex operations
////////////////////////////////////////////////////////////////////////////////

// Complex multiplication
static __device__ __host__ inline Complex ComplexMul(Complex a, Complex b) {
  Complex c;
  c.x = a.x * b.x - a.y * b.y;
  c.y = a.x * b.y + a.y * b.x;
  return c;
}

/*

std::printf("Signal array:\n");
    for (int i = 0; i < 5; i++) {
        std::printf("%f + %fj\n", hc_signal[i].x, hc_signal[i].y);
    }
    std::printf("=====\n");

std::printf("Filter array:\n");
    for (int i = 0; i < 5; i++) {
        std::printf("%f + %fj\n", hc_filter[i].x, hc_filter[i].y);
    }
    std::printf("=====\n");


std::printf("Host complex output array:\n");
    for (int i = 0; i < 5; i++) {
        std::printf("%f + %fj\n", hc_output[i].x, hc_output[i].y);
    }
    std::printf("=====\n");

std::printf("Host real output array:\n");
    for (int i = 0; i < 5; i++) {
        std::printf("%f\n", h_output[i]);
    }
    std::printf("=====\n");

*/