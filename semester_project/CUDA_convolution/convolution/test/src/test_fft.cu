/* TODO: Implement Callbacks

https://developer.nvidia.com/blog/cuda-pro-tip-use-cufft-callbacks-custom-data-processing/


*/

/* Include C++ stuff */
#include <complex>
#include <string.h>
#include <cstdio>
#include <cstdlib>

/* Include my stuff */
#include <convolution.h>
#include <Timer.hpp>

/* Include CUDA stuff */
#include <hip/hip_runtime.h>
#include <hipfft/hipfftXt.h>

#define N 1024
#define K 1024
#define BATCH_SIZE 1

using namespace std;

int main() {
    hipfftHandle plan1;
    hipfftHandle plan2;
    hipStream_t stream = NULL;

    int FFT_SIZE = next_power_of_2(N + K - 1);
    
    bool file_status = false;
    string signal_file_name =
        "/home/jeff/code/CPE613/semester_project/test_data_gold/arr1_1024.txt";
    string filter_file_name =
        "/home/jeff/code/CPE613/semester_project/test_data_gold/arr2_1024.txt";
    const char *output_file_name =
        "/home/jeff/code/CPE613/semester_project/test_data/cuda_fft_1024.txt";

    // Initialize the signal
    vector<hipfftComplex> h_signal(FFT_SIZE, hipfftComplex{0});
    vector<hipfftComplex> h_signal_fft(FFT_SIZE, hipfftComplex{0});
    
    file_status = read_file_into_vector(signal_file_name, h_signal);
    if (file_status == false) return EXIT_FAILURE;

    hipfftComplex *d_signal = nullptr;
    hipfftComplex *d_signal_fft = nullptr;
    hipMalloc(reinterpret_cast<void **>(&d_signal),
                sizeof(hipfftComplex) * h_signal.size());
    hipMalloc(reinterpret_cast<void **>(&d_signal_fft),
                sizeof(hipfftComplex) * h_signal_fft.size());

    // Initialize the filter
    vector<hipfftComplex> h_filter(FFT_SIZE, hipfftComplex{0});
    vector<hipfftComplex> h_filter_fft(FFT_SIZE, hipfftComplex{0});

    file_status = read_file_into_vector(filter_file_name, h_filter);
    if (file_status == false) return EXIT_FAILURE;
    
    hipfftComplex *d_filter = nullptr;
    hipfftComplex *d_filter_fft = nullptr;
    hipMalloc(reinterpret_cast<void **>(&d_filter),
                sizeof(hipfftComplex) * h_filter.size());
    hipMalloc(reinterpret_cast<void **>(&d_filter_fft),
                sizeof(hipfftComplex) * h_filter_fft.size());

    // Initialize the product
    vector<hipfftComplex> h_convolved_result(FFT_SIZE, hipfftComplex{0});
    
    hipfftComplex *d_convolved_fft = nullptr;
    hipfftComplex *d_product_fft = nullptr;

    hipMalloc(reinterpret_cast<void **>(&d_convolved_fft),
                sizeof(hipfftComplex) * FFT_SIZE);
    hipMalloc(reinterpret_cast<void **>(&d_product_fft),
                sizeof(hipfftComplex) * FFT_SIZE);

    checkCudaErrors(
        hipMemcpyAsync(
            d_signal, h_signal.data(),
            sizeof(hipfftComplex) * h_signal.size(),
            hipMemcpyHostToDevice,
            stream
        )
    );

    checkCudaErrors(
        hipMemcpyAsync(
            d_filter, h_filter.data(),
            sizeof(hipfftComplex) * h_filter.size(),
            hipMemcpyHostToDevice,
            stream
        )
    );

    hipfftCreate(&plan1);
    hipfftPlan1d(&plan1, FFT_SIZE, HIPFFT_C2C, BATCH_SIZE);
    //hipStreamCreateWithFlags(&stream, hipStreamNonBlocking);
    //hipfftSetStream(plan1, stream);

    // Process signal    
    hipfftExecC2C(plan1, d_signal, d_signal_fft, HIPFFT_FORWARD);

    // Process filter
    hipfftExecC2C(plan1, d_filter, d_filter_fft, HIPFFT_FORWARD);

    //hipStreamSynchronize(stream); // force CPU thread to wait

    checkCudaErrors(hipGetLastError());

    // dumpGPUDataToFile(d_signal_fft, {FFT_SIZE,1}, "cuda-fft-signal.txt");
    // dumpGPUDataToFile(d_filter_fft, {FFT_SIZE,1}, "cuda-fft-filter.txt");

    // Multiplication section
    complexMulAndScaleGPU(
        d_signal_fft,
        d_filter_fft,
        d_product_fft,
        FFT_SIZE
    );

    // dumpGPUDataToFile(d_product_fft, {FFT_SIZE,1}, "test.txt");

    // Perform inverse
    hipfftCreate(&plan2);
    hipfftPlan1d(&plan2, FFT_SIZE, HIPFFT_C2C, BATCH_SIZE);

    // Execute the inverse FFT on the result
    hipfftExecC2C(plan2, d_product_fft, d_convolved_fft, HIPFFT_BACKWARD);
    
    hipMemcpyAsync(
        h_convolved_result.data(), d_convolved_fft,
        sizeof(hipfftComplex) * FFT_SIZE,
        hipMemcpyDeviceToHost,
        stream
    );

    hipStreamSynchronize(stream); // force CPU thread to wait

    FILE* filePtr = fopen(output_file_name, "w");
    float tmp;
    for (int i = 0; i < FFT_SIZE - 1; i++) {
        tmp = complex_to_float(h_convolved_result[i]);
        typeSpecificfprintf(filePtr, tmp);
    }
    fclose(filePtr);
    
    /* free resources */
    hipFree(d_signal);
    hipFree(d_signal_fft);
    hipFree(d_filter);
    hipFree(d_filter_fft);
    hipFree(d_product_fft);
    hipFree(d_convolved_fft);

    hipfftDestroy(plan1);
    hipfftDestroy(plan2);
    hipStreamDestroy(stream);

    hipDeviceReset();

    return EXIT_SUCCESS;
}