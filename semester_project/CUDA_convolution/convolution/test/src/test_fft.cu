/* Include C++ stuff */
#include <complex>
#include <string.h>
#include <cstdio>
#include <cstdlib>

/* Include my stuff */
#include <convolution.h>
#include <Timer.hpp>

/* Include CUDA stuff */
#include <hip/hip_runtime.h>
#include <hipfft/hipfftXt.h>

#define N 1024
#define K 1024
#define BATCH_SIZE 1

using namespace std;

int main() {
    hipfftHandle plan1;
    hipfftHandle plan2;
    hipfftHandle plan3;
    hipStream_t stream = NULL;

    using scalar_type = float;
    using input_type = scalar_type;
    using output_type = complex<scalar_type>;

    vector<input_type> h_signal(N, 0);
    vector<hipfftComplex> h_signal_fft(static_cast<int>((N / 2 + 1)));
    input_type *d_signal = nullptr;
    hipfftComplex *d_signal_fft = nullptr;

    hipMalloc(reinterpret_cast<void **>(&d_signal),
                sizeof(input_type) * h_signal.size());
    hipMalloc(reinterpret_cast<void **>(&d_signal_fft),
                sizeof(output_type) * h_signal_fft.size());

    vector<input_type> h_filter(K, 0);
    vector<hipfftComplex> h_filter_fft(static_cast<int>((K / 2 + 1)));
    input_type *d_filter = nullptr;
    hipfftComplex *d_filter_fft = nullptr;

    hipMalloc(reinterpret_cast<void **>(&d_filter),
                sizeof(input_type) * h_filter.size());
    hipMalloc(reinterpret_cast<void **>(&d_filter_fft),
                sizeof(output_type) * h_filter_fft.size());

    vector<hipfftComplex> h_product_fft(static_cast<int>((N / 2 + 1)));
    hipfftComplex *d_product_fft = nullptr;
    hipMalloc(reinterpret_cast<void **>(&d_product_fft),
                sizeof(output_type) * static_cast<int>((N / 2 + 1)));

    hipfftReal* d_result = nullptr;
    hipMalloc(reinterpret_cast<void **>(&d_result), sizeof(hipfftReal) * N);

    vector<input_type> h_result(N, 0);
    
    hipStreamSynchronize(stream); // force CPU thread to wait

    // Prepare to read signal and filter information from files
    string signal_file_name =
        "/home/jeff/code/CPE613/semester_project/test_data_gold/arr1_1024.txt";
    string filter_file_name =
        "/home/jeff/code/CPE613/semester_project/test_data_gold/arr2_1024.txt";
    // const char *output_file_name =
    //     "/home/jeff/code/CPE613/semester_project/test_data/cuda_fft_1024.txt";

    bool file_status = false;
    file_status = read_file_into_vector(signal_file_name, h_signal);
    if (file_status == false) return 1;
    file_status = read_file_into_vector(filter_file_name, h_filter);
    if (file_status == false) return 1;

    printf("Signal array:\n");
    int x = 0;
    for (auto &i : h_signal) {
        printf("%d : %f\n", x, i);
        x++;
    }
    printf("=====\n");

    printf("Filter array:\n");
    int y = 0;
    for (auto &i : h_filter) {
        printf("%d : %f\n", y, i);
        y++;
    }
    printf("=====\n");

    return 0;

    hipMemcpyAsync(d_signal, h_signal.data(),
                    sizeof(input_type) * h_signal.size(),
                    hipMemcpyHostToDevice,
                    stream
                );

    hipMemcpyAsync(d_filter, h_signal.data(),
                    sizeof(input_type) * h_filter.size(),
                    hipMemcpyHostToDevice,
                    stream
                );

    hipfftCreate(&plan1);
    hipfftPlan1d(&plan1, h_signal.size(), HIPFFT_R2C, N);

    hipStreamCreateWithFlags(&stream, hipStreamNonBlocking);
    hipfftSetStream(plan1, stream);

    hipfftExecR2C(plan1, d_signal, d_signal_fft);
    hipfftDestroy(plan1);

    hipfftCreate(&plan2);
    hipfftPlan1d(&plan2, h_filter.size(), HIPFFT_R2C, K);

    hipStreamCreateWithFlags(&stream, hipStreamNonBlocking);
    hipfftSetStream(plan2, stream);

    hipfftExecR2C(plan2, d_filter, d_filter_fft);
    hipfftDestroy(plan2);

    hipStreamSynchronize(stream); // force CPU thread to wait

    // Multiplication section
    complexMulGPU(
        d_signal_fft,
        d_filter_fft,
        d_product_fft,
        static_cast<int>((N / 2 + 1))
    );

    // Perform inverse
    hipfftCreate(&plan3);
    hipfftPlan1d(&plan3, h_product_fft.size(), HIPFFT_C2R, BATCH_SIZE);

    // Execute the inverse FFT on the result
    hipfftExecC2R(plan3, d_product_fft, (hipfftReal*)d_result);
    
    hipMemcpyAsync(h_result.data(), d_result, sizeof(hipfftReal) * N,
                                 hipMemcpyDeviceToHost, stream);

    printf("Real result array:\n");
    for (auto &i : h_result) {
        printf("%f\n", i);
    }
    printf("=====\n");
    
    /* free resources */
    hipFree(d_signal);
    hipFree(d_signal_fft);
    hipFree(d_filter);
    hipFree(d_filter_fft);
    hipFree(d_result);
    hipFree(d_product_fft);

    hipStreamDestroy(stream);

    hipDeviceReset();

    return EXIT_SUCCESS;
}

/*

std::printf("Signal array:\n");
    for (int i = 0; i < 5; i++) {
        std::printf("%f + %fj\n", hc_signal[i].x, hc_signal[i].y);
    }
    std::printf("=====\n");

std::printf("Filter array:\n");
    for (int i = 0; i < 5; i++) {
        std::printf("%f + %fj\n", hc_filter[i].x, hc_filter[i].y);
    }
    std::printf("=====\n");


std::printf("Host complex output array:\n");
    for (int i = 0; i < 5; i++) {
        std::printf("%f + %fj\n", hc_output[i].x, hc_output[i].y);
    }
    std::printf("=====\n");

std::printf("Host real output array:\n");
    for (int i = 0; i < 5; i++) {
        std::printf("%f\n", h_output[i]);
    }
    std::printf("=====\n");

*/