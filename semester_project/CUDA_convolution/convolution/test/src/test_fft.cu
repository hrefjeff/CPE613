/* TODO: Implement Callbacks

https://developer.nvidia.com/blog/cuda-pro-tip-use-cufft-callbacks-custom-data-processing/

*/

/* Include C++ stuff */
#include <complex>
#include <string.h>
#include <cstdio>
#include <cstdlib>

/* Include CUDA stuff */
#include <hip/hip_runtime.h>
#include <hipfft/hipfftXt.h>

/* Include my stuff */
#include <convolution.h>
#include <Timer.hpp>

#define N 4096
#define K 4096
#define BATCH_SIZE 1

using namespace std;

int main() {
    hipfftHandle plan;
    hipStream_t stream = NULL;

    int FFT_SIZE = next_power_of_2(N + K - 1);
    
    bool file_status = false;
    string signal_file_name =
        "/home/jeff/code/CPE613/semester_project/test_data_gold/arr1_4096.txt";
    string filter_file_name =
        "/home/jeff/code/CPE613/semester_project/test_data_gold/arr2_4096.txt";
    const char *output_file_name =
        "/home/jeff/code/CPE613/semester_project/test_data/cuda_fft_4096.txt";

    // Initialize the signal

    // host signal initialization
    vector<hipfftComplex> h_signal(FFT_SIZE, hipfftComplex{0});
    
    file_status = read_file_into_vector(signal_file_name, h_signal);
    if (file_status == false) return EXIT_FAILURE;

    // device signal initialization
    hipfftComplex *d_signal = nullptr;
    checkCudaErrors(
        hipMalloc(
            reinterpret_cast<void **>(&d_signal),
            sizeof(hipfftComplex) * h_signal.size()
        )
    );

    // Initialize the filter

    // host filter initialization
    vector<hipfftComplex> h_filter(FFT_SIZE, hipfftComplex{0});

    file_status = read_file_into_vector(filter_file_name, h_filter);
    if (file_status == false) return EXIT_FAILURE;

    // device filter initialization    
    hipfftComplex *d_filter = nullptr;
    checkCudaErrors(
        hipMalloc(
            reinterpret_cast<void **>(&d_filter),
            sizeof(hipfftComplex) * h_filter.size()
        )
    );

    // Initialize the result

    // host result initialization
    vector<hipfftComplex> h_convolved_result(FFT_SIZE, hipfftComplex{0});
    
    // device result inintialization
    hipfftComplex *d_convolved_fft = nullptr;
    hipfftComplex *d_product_fft = nullptr;

    checkCudaErrors(
        hipMalloc(
            reinterpret_cast<void **>(&d_convolved_fft),
            sizeof(hipfftComplex) * h_convolved_result.size()
        )
    );
    checkCudaErrors(
        hipMalloc(
            reinterpret_cast<void **>(&d_product_fft),
            sizeof(hipfftComplex) * h_convolved_result.size()
        )
    );

    // Copy host data to device
    checkCudaErrors(
        hipMemcpyAsync(
            d_signal, h_signal.data(),
            sizeof(hipfftComplex) * h_signal.size(),
            hipMemcpyHostToDevice,
            stream
        )
    );

    checkCudaErrors(
        hipMemcpyAsync(
            d_filter, h_filter.data(),
            sizeof(hipfftComplex) * h_filter.size(),
            hipMemcpyHostToDevice,
            stream
        )
    );

    Timer timer;
    timer.start();
    hipfftCreate(&plan);
    hipfftPlan1d(&plan, FFT_SIZE, HIPFFT_C2C, BATCH_SIZE);

    // Process signal    
    hipfftExecC2C(plan, d_signal, d_signal, HIPFFT_FORWARD);

    // Process filter
    hipfftExecC2C(plan, d_filter, d_filter, HIPFFT_FORWARD);

    // Perform Discrete Fourier Transform by multiplying the signals in frequency form
    complexMulAndScaleGPU(
        d_signal,
        d_filter,
        d_product_fft,
        FFT_SIZE
    );

    // Execute the inverse FFT on the result
    hipfftExecC2C(plan, d_product_fft, d_product_fft, HIPFFT_BACKWARD);
    timer.stop();

    
    checkCudaErrors(
        hipMemcpyAsync(
            h_convolved_result.data(), d_product_fft,
            sizeof(hipfftComplex) * FFT_SIZE,
            hipMemcpyDeviceToHost,
            stream
        )
    );
    double elapsedTime_ms = timer.elapsedTime_ms();

    printf (
    "\n- Elapsed Time:             %20.16e Ms\n\n",
        elapsedTime_ms / 1.0e3
    );

    file_status = write_results_to_file(output_file_name, h_convolved_result);
    if (file_status == false) return EXIT_FAILURE;
    
    /* free resources */
    hipFree(d_signal);
    hipFree(d_filter);
    hipFree(d_product_fft);
    hipFree(d_convolved_fft);

    hipfftDestroy(plan);
    hipStreamDestroy(stream);

    return EXIT_SUCCESS;
}