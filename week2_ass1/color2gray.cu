#include "hip/hip_runtime.h"
#include <cstdlib>
#include <iostream>
#include <opencv2/opencv.hpp>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>

using namespace cv;
using namespace std;

__global__ 
void rgb2gray_kernel (
    unsigned char* red_d,
    unsigned char* green_d,
    unsigned char* blue_d,
    unsigned char* gray_d,
    int width,
    int height
) {

    for (
        int rowIdx = threadIdx.y + blockIdx.y * blockDim.y;
        rowIdx < height;
        rowIdx += blockDim.y * gridDim.y
    ) {
        for (
            int colIdx = threadIdx.y + blockIdx.y * blockDim.y;
            colIdx < width;
            colIdx += blockDim.x * gridDim.x
        ) {
            int offset = rowIdx * width + colIdx;
            gray_d[offset] = (unsigned char)(
                (float)red_d[offset] * 3.0 / 10.0 +
                (float)green_d[offset] * 6.0 / 10.0 +
                (float)blue_d[offset] * 1.0 / 10.0
            );
        }
    }

}

void device_rgb2grayscale (
    unsigned char * deviceRed,
    unsigned char * deviceGreen,
    unsigned char * deviceBlue,
    unsigned char * deviceGray,
    int numRows,
    int numCols
){
    int blockWidth = 16;

    dim3 blockSize(blockWidth, blockWidth);

    // Set our block size and threads per thread block
    const int blockWidth = 16;

    // Set up kernel launch parameters, so we can create grid/blocks
    dim3 numThreadsPerBlock(blockWidth, blockWidth);
    dim3 numBlocks(
        (numCols + numThreadsPerBlock.x - 1)/numThreadsPerBlock.x,
        (numRows + numThreadsPerBlock.y - 1)/numThreadsPerBlock.y
    );

    // Perform CUDA computations on deviceMatrix, Launch Kernel
    rgb2gray_kernel<<<
        numBlocks,
        numThreadsPerBlock
    >>>(
        deviceRed,
        deviceGreen,
        deviceBlue,
        deviceGray,
        numCols,
        numRows
    );

    checkCudaErrors(
        hipGetLastError()
    );

}


int main() {

    Mat img = imread("thethreeamigos.jpeg", IMREAD_COLOR);
    //imshow("Goat!", img);

    // Set our problem size
    const int WIDTH = 810;
    const int HEIGHT = 456;
    const int TOTAL_SIZE = WIDTH * HEIGHT;

    // Allocate memory in host RAM
    std::vector<unsigned char> hostRed(TOTAL_SIZE);
    std::vector<unsigned char> hostGreen(TOTAL_SIZE);
    std::vector<unsigned char> hostBlue(TOTAL_SIZE);
    std::vector<unsigned char> hostGray(TOTAL_SIZE);

    // Fill the host matrices with data
    Mat greyMat(img.rows, img.cols, CV_8UC1, Scalar(0));
    for (int rowIdx = 0; rowIdx < HEIGHT; ++rowIdx) {
        for (int colIdx = 0; colIdx < WIDTH; ++colIdx) {
            auto & vec = img.at<cv::Vec<uchar, 3>>(rowIdx, colIdx);
            int offset = rowIdx * WIDTH + colIdx;
            hostBlue[offset] = vec[0]; 
            hostGreen[offset] = vec[1]; 
            hostRed[offset] = vec[2];
        }
    }

    // Allocate memory space on the device
    unsigned char * deviceRed = nullptr;
    unsigned char * deviceGreen = nullptr;
    unsigned char * deviceBlue = nullptr;
    unsigned char * deviceGray = nullptr;
    size_t byteSize = HEIGHT * WIDTH * sizeof(unsigned char);
    checkCudaErrors(hipMalloc(&deviceRed,byteSize));
    checkCudaErrors(hipMalloc(&deviceGreen,byteSize));
    checkCudaErrors(hipMalloc(&deviceBlue,byteSize));
    checkCudaErrors(hipMalloc(&deviceGray,byteSize));

    // Upload data to device
    checkCudaErrors(
        hipMemcpy (deviceRed, hostRed.data(),byteSize,hipMemcpyHostToDevice)
    );
    checkCudaErrors(
        hipMemcpy (deviceGreen, hostGreen.data(),byteSize,hipMemcpyHostToDevice)
    );
    checkCudaErrors(
        hipMemcpy (deviceBlue, hostBlue.data(),byteSize,hipMemcpyHostToDevice)
    );

    device_rgb2grayscale (
        deviceRed,
        deviceGreen,
        deviceBlue,
        deviceGray,
        HEIGHT,
        WIDTH
    );

    // Copy result from device to host
    checkCudaErrors(
        hipMemcpy (hostGray.data(), deviceGray,byteSize,hipMemcpyDeviceToHost)
    );

    // Copy result from gray matrix into matlab OpenCV input array format
    for (int rowIdx = 0; rowIdx < HEIGHT; ++rowIdx) {
    for (int colIdx = 0; colIdx < WIDTH; ++colIdx)
      greyMat.at<uchar>(rowIdx, colIdx) = hostGray[(rowIdx*WIDTH)+colIdx];
    }

    // Write img to gray.jpg
    imwrite("grayboiz.jpg", greyMat);

    // Free memory
    hipFree(deviceRed);
    hipFree(deviceGreen);
    hipFree(deviceBlue);
    hipFree(deviceGray);

    printf("Made it to the end!\n");

    return 0;
}
