#include "hip/hip_runtime.h"
#include <cstdlib>
#include <cassert>
#include <iostream>

using namespace std;

__global__ 
void colorToGrayScaleConversion(unsigned char *Pout, unsigned char *Pin, int width, int height) {

    int col = blockIdx.x * blockDim.x + threadIdx.x;
    int row = blockIdx.y * blockDim.y + threadIdx.y;

    if (col < width && row < height) {
        // Get 1D offset for the grayscale image
        int grayOffset = row*width + col;

        // One can think of the RGB image having CHANNEL
        // times more columns than the gray scale image
        int rgbOffset = grayOffset*3; // 3 channels
        unsigned char r = Pin[rgbOffset    ];
        unsigned char g = Pin[rgbOffset + 1];
        unsigned char b = Pin[rgbOffset + 2];

        // Perform the rescaling and store it
        // We multiply by floating point constants
        Pout[grayOffset] = 0.21f*r + 0.71f*g+ 0.07f*b;
    }
}

int main() {

    // Set our problem size
    const int WIDTH = 810;
    const int HEIGHT = 456;
    int *hostMatrix;
    int *deviceMatrix;
    // TODO: Find out what Pin and Pout are
    
    // Allocate memory on the host
    hipHostMalloc(&hostMatrix, row * col * sizeof(int));

    // Fill the host matrix with data
    for (int i = 0; i < row; i++) {
        for (int j = 0; j < col; j++) {
            hostMatrix[i * col + j] = /* TODO: GET VALUES FROM RGB MATRIX */;
        }
    }

    // Allocate memory on the device
    hipMalloc(&deviceMatrix, row * col * sizeof(int));

    // Set our block size and threads per thread block
    const int THREADS = 32;

    // Set up kernel launch parameters, so we can create grid/blocks
    dim3 numThreadsPerBlock(THREADS, THREADS);
    dim3 numBlocks( (WIDTH + numThreadsPerBlock.x - 1)/numThreadsPerBlock.x,
                    (HEIGHT + numThreadsPerBlock.y - 1)/numThreadsPerBlock.y);

    

    // Copy data from host to device
    hipMemcpy(deviceMatrix, hostMatrix, row * col * sizeof(int), hipMemcpyHostToDevice);

    // Perform CUDA computations on deviceMatrix
    // Launch our kernel
    colorToGrayScaleConversion<<<numBlocks, numThreadsPerBlock>>>(/*pic in*/, /*pic out*/, HEIGHT, WIDTH);

    // Free memory
    hipFree(deviceMatrix);
    hipHostFree(hostMatrix);

    return 0;
}
