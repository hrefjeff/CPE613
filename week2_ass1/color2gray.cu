#include "hip/hip_runtime.h"
#include <iostream>
#include <opencv2/opencv.hpp>

using namespace cv;
using namespace std;

__global__ 
void rgb2gray_kernel(unsigned char* red,unsigned char* green, unsigned char* blue, 
                    unsigned char* gray, int width, int height) {

    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;

    // Boundary check
    if (col < width && row < height) {
        // Get 1D offset for the grayscale image
        int pixelIndex = (row*width) + col;

        // Convert the pixel
        gray[pixelIndex] = (red[pixelIndex]*3.0/10.0) + (green[pixelIndex]*6.0/10.0) + (blue[pixelIndex]*1.0/10.0);
    }
}

int main() {

    Mat img = imread("thethreeamigos.jpeg", IMREAD_COLOR);
    // imshow("Goat!", img);

    // Set our problem size
    const int WIDTH = 810;
    const int HEIGHT = 456;
    const int TOTAL_SIZE = WIDTH * HEIGHT;

    // Allocate memory in host RAM
    unsigned char *h_red, *h_green, *h_blue, *h_gray;
    hipHostMalloc((void **) &h_red, sizeof(char)*TOTAL_SIZE);
    hipHostMalloc((void **) &h_green, sizeof(char)*TOTAL_SIZE);
    hipHostMalloc((void **) &h_blue, sizeof(char)*TOTAL_SIZE);
    hipHostMalloc((void **) &h_gray, sizeof(char)*TOTAL_SIZE);

    // Fill the host matrices with data
    Mat greyMat(img.rows, img.cols, CV_8UC1, Scalar(0));
    for (int rowIdx = 0; rowIdx < img.rows; ++rowIdx) {
        for (int colIdx = 0; colIdx < img.cols; ++colIdx) {
            auto & vec = img.at<cv::Vec<uchar, 3>>(rowIdx, colIdx);
            h_blue[rowIdx+colIdx] = vec[0]; 
            h_green[rowIdx+colIdx] = vec[1]; 
            h_red[rowIdx+colIdx] = vec[2];
        }
    }

    // Allocate memory space on the device 
    unsigned char *d_red, *d_green, *d_blue, *d_gray;
    hipMalloc((void **) &d_red, sizeof(char)*TOTAL_SIZE);
    hipMalloc((void **) &d_green, sizeof(char)*TOTAL_SIZE);
    hipMalloc((void **) &d_blue, sizeof(char)*TOTAL_SIZE);
    hipMalloc((void **) &d_gray, sizeof(char)*TOTAL_SIZE);

    // Copy matrices from host to device memory
    hipMemcpy(d_red, h_red, sizeof(char)*TOTAL_SIZE, hipMemcpyHostToDevice);
    hipMemcpy(d_green, h_green, sizeof(char)*TOTAL_SIZE, hipMemcpyHostToDevice);
    hipMemcpy(d_blue, h_blue, sizeof(char)*TOTAL_SIZE, hipMemcpyHostToDevice);

    // Set our block size and threads per thread block
    const int THREADS = 32;

    // Set up kernel launch parameters, so we can create grid/blocks
    dim3 numThreadsPerBlock(THREADS, THREADS);
    dim3 numBlocks( (WIDTH + numThreadsPerBlock.x - 1)/numThreadsPerBlock.x,
                    (HEIGHT + numThreadsPerBlock.y - 1)/numThreadsPerBlock.y);

    // Perform CUDA computations on deviceMatrix, Launch Kernel
    rgb2gray_kernel<<<numBlocks, numThreadsPerBlock>>>(d_red, d_green, d_blue, d_gray, HEIGHT, WIDTH);

    // Copy result from device to host
    hipMemcpy(d_gray, h_gray, TOTAL_SIZE, hipMemcpyDeviceToHost);

    // Copy result from gray matrix into matlab OpenCV input array format
    for (int rowIdx = 0; rowIdx < HEIGHT; ++rowIdx) {
    for (int colIdx = 0; colIdx < WIDTH; ++colIdx)
      greyMat.at<uchar>(rowIdx, colIdx) = h_gray[rowIdx + colIdx];
    }

    // Write img to gray.jpg
    imwrite("grayboiz.jpg", greyMat);

    // Free memory
    hipFree(d_red);
    hipFree(d_green);
    hipFree(d_blue);
    hipFree(d_gray);
    hipHostFree(h_red);
    hipHostFree(h_green);
    hipHostFree(h_blue);
    hipHostFree(h_gray);

    return 0;
}
