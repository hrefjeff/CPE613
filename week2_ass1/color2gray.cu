
#include <hip/hip_runtime.h>
#include <cstdlib>
#include <cassert>
#include <iostream>

using namespace std;

__global__ 
void rgb2gray_kernel(unsigned char* red,unsigned char* green, unsigned char* blue, 
                    unsigned char* gray, int width, int height) {

    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;

    // Boundary check
    if (col < width && row < height) {
        // Get 1D offset for the grayscale image
        int pixelIndex = row*width + col;

        // Convert the pixel
        gray[pixelIndex] = red[pixelIndex]*3/10 + green[pixelIndex]*6/10 + blue[pixelIndex]*1/10;
    }
}

int main() {

    // Set our problem size
    const int WIDTH = 810;
    const int HEIGHT = 456;
    const int TOTAL_SIZE = WIDTH * HEIGHT;
    unsigned char *red_h, *green_h, *blue_h, *gray_h;
    unsigned char *red_d, *green_d, *blue_d, *gray_d;
    
    // Allocate memory on the host
    hipHostMalloc((void**)&red_h, TOTAL_SIZE, hipHostMallocDefault);
    hipHostMalloc((void**)&green_h, TOTAL_SIZE, hipHostMallocDefault);
    hipHostMalloc((void**)&blue_h, TOTAL_SIZE, hipHostMallocDefault);
    hipHostMalloc((void**)&gray_h, TOTAL_SIZE, hipHostMallocDefault);

    // Fill the host matrix with data
    FILE *red_file = fopen("reds.txt", "r");
    FILE *green_file = fopen("greens.txt", "r");
    FILE *blue_file = fopen("blues.txt", "r");
    if (red_file == NULL || green_file == NULL || blue_file == NULL) {
        printf("Error opening file\n");
        return 1;
    }
    
    for (int i = 0; i < HEIGHT; i++) {
        for (int j = 0; j < WIDTH; j++) {
            fscanf(red_file, "%d", &red_h[i+j]);
            fscanf(green_file, "%d", &green_h[i+j]);
            fscanf(blue_file, "%d", &blue_h[i+j]);
        }
    }

    fclose(red_file);
    fclose(green_file);
    fclose(blue_file);

    // Allocate memory on the device
    hipMalloc(&red_d, WIDTH * HEIGHT * sizeof(int));
    hipMalloc(&green_d, TOTAL_SIZE);
    hipMalloc(&blue_d, TOTAL_SIZE);
    hipMalloc(&gray_d, TOTAL_SIZE);

    // Set our block size and threads per thread block
    const int THREADS = 32;

    // Set up kernel launch parameters, so we can create grid/blocks
    dim3 numThreadsPerBlock(THREADS, THREADS);
    dim3 numBlocks( (WIDTH + numThreadsPerBlock.x - 1)/numThreadsPerBlock.x,
                    (HEIGHT + numThreadsPerBlock.y - 1)/numThreadsPerBlock.y);

    // Copy data from host to device
    hipMemcpy(red_d, red_h, TOTAL_SIZE, hipMemcpyHostToDevice);
    hipMemcpy(green_d, green_h, TOTAL_SIZE, hipMemcpyHostToDevice);
    hipMemcpy(blue_d, blue_h, TOTAL_SIZE, hipMemcpyHostToDevice);

    // Perform CUDA computations on deviceMatrix
    // Launch our kernel
    rgb2gray_kernel<<<numBlocks, numThreadsPerBlock>>>(red_d, green_d, blue_d, gray_d, HEIGHT, WIDTH);

    // Free memory
    hipFree(red_d);
    hipFree(green_d);
    hipFree(blue_d);
    hipFree(gray_d);
    hipHostFree(red_h);
    hipHostFree(green_h);
    hipHostFree(blue_h);
    hipHostFree(gray_h);

    return 0;
}
