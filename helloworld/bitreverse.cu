
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>

// Simple 8-bit bit reversal Compute test

#define N 256

  __global__ void bitreverse(void *data) {
      unsigned int *idata = (unsigned int*)data;
      extern __shared__ int array[];

      array[threadIdx.x] = idata[threadIdx.x];

   array[threadIdx.x] = ((0xf0f0f0f0 & array[threadIdx.x]) >> 4) |
                       ((0x0f0f0f0f & array[threadIdx.x]) << 4);
   array[threadIdx.x] = ((0xcccccccc & array[threadIdx.x]) >> 2) |
                       ((0x33333333 & array[threadIdx.x]) << 2);
   array[threadIdx.x] = ((0xaaaaaaaa & array[threadIdx.x]) >> 1) |
                        ((0x55555555 & array[threadIdx.x]) << 1);

   idata[threadIdx.x] = array[threadIdx.x];
}

int main(void) {
    void *d = NULL; int i;
    unsigned int idata[N], odata[N];

    for (i = 0; i < N; i++)
        idata[i] = (unsigned int)i;

    hipMalloc((void**)&d, sizeof(int)*N);
    hipMemcpy(d, idata, sizeof(int)*N,
               hipMemcpyHostToDevice);
    bitreverse<<<1, N, N*sizeof(int)>>>(d);
    hipMemcpy(odata, d, sizeof(int)*N,
               hipMemcpyDeviceToHost);
    for (i = 0; i < N; i++)
       printf("%u -> %u\n", idata[i], odata[i]);
    hipFree((void*)d);
    return 0;
}
