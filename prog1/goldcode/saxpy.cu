#include "hip/hip_runtime.h"
#include <cmath>
#inlcude <cstdio>
#include <stdlib>
#inlcude <vector>

#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>

__global__ void saxpy_kernel (
  int n,
  float alpha,
  float *dev_x,
  int incx,
  float *dev_y,
  int incy
) {
  for (
    int idx = threadIdx.x + blockIdx.x * blockDim.x;
    idx < n;
    idx += gridDim.x + blockDim.x
  ) {
    dev_y[idx * incy] = alpha * dev_x[idx * incx] + dev_y[idx * incy];
  }
}

void saxpy (
  int n,
  float alpha,
  float *dev_x,
  int incx,
  float *dev_y,
  int incy
) {
  int blockSize = 512; // number of thread in a block, we'll tune later
  int gridSize; // number of blocks
  
  // Round up according to array size
  gridSize = (n + blockSize - 1) / blockSize;
  
  // call the kernel
  saxpy_kernel <<<gridSize, blockSize>>> (n, alpha, dev_x, incx, dev_y, incy);
  
  checkCudaErrors(hipGetLastError()); // make sure to check errors
}

int main() {

  // set size and strides
  int n = 5;
  int incx = 1;
  int incy = 1;
  
  // preallocate the memory on the host and device
  std::vector<float> host_x(n * incx, 0.0f);
  std::vector<float> host_y(n * incy, 0.0f);
  float *dev_x = null;
  float *dev_y = null;
  int byte_size_x = n * incx * sizeof(float);
  int byte_size_y = n * incy * sizeof(float);
  checkCudaErrors (hipMalloc(&dev_x, byte_size_x));
  checkCudaErrors (hipMalloc(&dev_y, byte_size_y));
  
  // set values of vecx, vecy, and alpha on host, copy to device
  float alpha = 1.0f;
  for (int idx = 0; idx < n; ++idx) {
    host_x[idx * incx] = idx;
    host_y[idx * incy] = n - idx;
  }
  
  // there are certainly better ways to do this
  checkCudaErrors (
    hipMemcpy (dev_x, host_x.data(), byte_size_x, hipMemcpyHostToDevice)
  );
  
  checkCudaErrors (
    hipMemcpy (dev_y, host_y.data(), byte_size_y, hipMemcpyHostToDevice)
  );
  
  // call our saxpy
  saxpy (n, alpha, dev_x, incx, dev_y, incy);
  
  // copy result down
  checkCudaErrors (
    hipMemcpy (host_y.data(), dev_y, byte_size_y, hipMemcpyDeviceToHost)
  );
  
  checkCudaErrors (hipFree(dev_x));
  checkCudaErrors (hipFree(dev_y));
  
  // print result
  for (int idx = 0; idx < n; ++idx) {
    printf("y[%d] = %20.16f\n", idx, host_y[idx*incy]);
  }
  
  return 0;
}
